﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <array>
#include <vector>
#include <random>
#include <iomanip>

#define SIZE 1024

// Matrix is a 4x4 array of array of ints
#define ORDER 3
#define MAT_SIZE ORDER*ORDER

#define MATRIX_COMP_MUL(_i, _j, _MATA, _MATB, _DEST) \
	_DEST[_i * ORDER + _j] = 0; \
	for(size_t k = 0; k < ORDER; ++k) \
		_DEST[_i * ORDER + _j] += _MATA[_i * ORDER + k] * _MATB[k * ORDER + _j];

#define PRINT_MATRIX_ROW(_row) \
	std::cout << "| "; \
	for(size_t _i = 0; _i < ORDER; ++_i) \
		std::cout << std::setw(5) << std::right << (_row)[_i] << ' '; \
	std::cout << '|';

__global__ void matrixMultiplyKernel(const int* matAPtr, const int* matBPtr, int* matDestPtr) {
	int id = threadIdx.x * MAT_SIZE;

	const int* matA = &matAPtr[id];
	const int* matB = &matBPtr[id];
    int* matDest = &matDestPtr[id];

	for (size_t i = 0; i < ORDER; ++i) {
		for (size_t j = 0; j < ORDER; ++j) {
			MATRIX_COMP_MUL(i, j, matA, matB, matDest);
		}
	}
}

/* Generates a random matrix. */
void generateRandomMatrix(int* mat) {
	static std::default_random_engine engine;
	std::uniform_int_distribution<int> dist(1, 200);
	for (size_t i = 0; i < ORDER; ++i)
		for (size_t j = 0; j < ORDER; ++j)
			mat[i * ORDER + j] = dist(engine);
}

/* Generates a list of N Random Matrices. Value of N is the value of SIZE. */
static void generateRandomMatrices(int* dest) {
	for (size_t i = 0; i < SIZE; ++i) {
		generateRandomMatrix(&dest[i * MAT_SIZE]);
	}
}

// Pointers to the list of matrices in GPU.
int* matricesAGPU;
int* matricesBGPU;
int* matricesDestGPU;

void GPUCleanup() {
	hipFree(matricesAGPU);
	hipFree(matricesBGPU);
	hipFree(matricesDestGPU);
}

/* Multiply the given matrices in parallel using CUDA. The source and destination arrays must have atleast
   MAT_ORDER * MAT_ORDER * SIZE number of elements.
   Arguments:                                         
   a_mats: Pointer to the first element of a 3D array containing the values of the first matrix operands.
   b_mats: Pointer to the first element of a 3D array containing the values of the second matrix operands.
   dest_mats: Pointer to the first element of a 3D array containing the values of the result matrices.
*/
hipError_t matrixMultiplyWithCuda(const int * matricesA, const int * matricesB, int * matricesDest) {
	hipError_t status;
	status = hipSetDevice(0);
	if (status != hipSuccess) {
		std::cerr << "Error initializing CUDA! Please check your GPU!" << std::endl;
		return status;
	}

	constexpr int sz = ORDER * ORDER * SIZE;

	std::cout << "Allocating memory on GPU for data...\r";
	status = hipMalloc((void**)&matricesAGPU, sz * sizeof(int));
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error allocating memory on CUDA device!" << std::endl;
		return status;
	}

	status = hipMalloc((void**)&matricesBGPU, sz * sizeof(int));
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error allocating memory on CUDA device!" << std::endl;
		return status;
	}

	status = hipMalloc((void**)&matricesDestGPU, sz * sizeof(int));
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error allocating memory on CUDA device!" << std::endl;
		return status;
	}
	std::cout << "Successfully allocated memory on the GPU device!" << std::endl;

	std::cout << "Copying data to GPU for calculation...\r";
	status = hipMemcpy(matricesAGPU, matricesA, sz * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error copying memory from CUDA device!" << std::endl;
		return status;
	}
	status = hipMemcpy(matricesBGPU, matricesB, sz * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error copying memory from CUDA device!" << std::endl;
		return status;
	}
	std::cout << "Successfully copied all the data to the GPU device!" << std::endl;

	matrixMultiplyKernel<<<1, SIZE>>>(matricesAGPU, matricesBGPU, matricesDestGPU);

	std::cout << "Cleaning up..." << std::endl;

	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		GPUCleanup();
		std::cout << "Error synchronizing GPU! Debugging required." << std::endl;
		return status;
	}

	status = hipMemcpy((void*)matricesDest, matricesDestGPU, sz * sizeof(int), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error copying memory from CUDA device!" << std::endl;
		return status;
	}

	status = hipDeviceReset();
	if (status != hipSuccess) {
		GPUCleanup();
		std::cerr << "Error resetting on GPU! Debugging required." << std::endl;
		return status;
	}

	GPUCleanup();
	return status;
}

void printMatrix(const int* a, const int* b, const int* c) {
	char sep = ' ';
	char eq = ' ';
	for (size_t i = 0; i < ORDER; ++i) {
		if (i + 1 == ORDER / 2) {
			sep = 'x';
			eq = '=';
		}
		else {
			sep = ' ';
			eq = ' ';
		}

		PRINT_MATRIX_ROW(&a[i * ORDER]);
		std::cout << sep;
		PRINT_MATRIX_ROW(&b[i * ORDER]);
		std::cout << eq;
		PRINT_MATRIX_ROW(&c[i * ORDER]);
		std::cout << '\n';
	}
}

void printMatrices(const int* matricesA, const int* matricesB, const int* matricesC) {
	std::ios::sync_with_stdio(false);
	std::cout << std::endl;
	std::cout << std::setprecision(5);
	for (size_t i = 0; i < SIZE; ++i) {
		printMatrix(&matricesA[i * MAT_SIZE], &matricesB[i * MAT_SIZE], &matricesC[i * MAT_SIZE]);
		std::cout << '\n';
	}
	std::cout << std::flush;
	std::ios::sync_with_stdio(true);
}

int main() {
	std::cout << "CUDA Test made by Dot." << std::endl;
	std::cout << "CUDA Program to multiply " << SIZE << " number of " 
		<< ORDER << 'x' << ORDER
		<< " random matrices and print their result." << std::endl;

	int* matricesA = new int[MAT_SIZE * SIZE];
	int* matricesB = new int[MAT_SIZE * SIZE];
	int* matricesDest = new int[MAT_SIZE * SIZE];

	std::cout << "Generating random matrices...\r";
	generateRandomMatrices(matricesA);
	generateRandomMatrices(matricesB);
	std::cout << "Random matrices for use as operands have been generated!" << std::endl;

	std::cout << "Multiplying matrices using CUDA..." << std::endl;
	matrixMultiplyWithCuda(matricesA, matricesB, matricesDest);
	std::cout << "Successfully multiplied the matrices using CUDA! Printing results now..." << std::endl;

	printMatrices(matricesA, matricesB, matricesDest);

	std::cout << "Program successfully executed!" << std::endl;

	return 0;
}
